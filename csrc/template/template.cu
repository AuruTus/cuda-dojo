#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = (call); \
        if (err != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while(0)

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 10000;
    const int block_size = 256;
    const int grid_size = (N + block_size - 1) / block_size;

    // Host arrays
    int* h_a = new int[N];
    int* h_b = new int[N];
    int* h_c = new int[N];

    // Device arrays
    int* d_a, * d_b, * d_c;

    // Initialize host data
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate device memory with error checking
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(int)));

    // Copy to device
    CUDA_CHECK(hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel
    printf("Launching kernel with %d blocks, %d threads per block\n",
           grid_size, block_size);
    vectorAdd << <grid_size, block_size >> > (d_a, d_b, d_c, N);

    // Check kernel execution
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy back results
    CUDA_CHECK(hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost));

    // Verify results
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Error at index %d: %d != %d + %d\n",
                   i, h_c[i], h_a[i], h_b[i]);
            success = false;
            break;
        }
    }

    if (success) {
        printf("Kernel executed successfully!\n");
        printf("Sample result: %d + %d = %d\n", h_a[100], h_b[100], h_c[100]);
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}